#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include "cudamat_kernels.cuh"
#include "cudamat.cuh"

extern "C" {

/* ------------------------------ CUBLAS init/shutdown ------------------------------ */

inline bool check_cublas_error() {
    hipblasStatus_t status = cublasGetError();

    return status != HIPBLAS_STATUS_SUCCESS;
}

inline bool checkCUDAError() {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
        printf("%s\n", hipGetErrorString( err));
    return hipSuccess != err;
}

extern const char* get_last_cuda_error() {
    hipError_t err = hipGetLastError();

    return hipGetErrorString( err);
}

extern int cublas_init() {
    cublasInit();
    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

extern int cublas_shutdown() {
    cublasShutdown();
    hipDeviceReset();

    return 0;
}


extern int cuda_set_device(int deviceId) {
    hipSetDevice(deviceId);
    
    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int init_random(rnd_struct* rnd_state, int seed, char* cudamatpath) {
    unsigned int * host_mults;
    host_mults = (unsigned int*)malloc(NUM_RND_STREAMS * sizeof(unsigned int));
    FILE * pFile;

    pFile = fopen (cudamatpath,"r");

    for (int i = 0; i < NUM_RND_STREAMS; i++) {
        fscanf (pFile, "%u", &host_mults[i]);
    }
    fclose (pFile);

    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned int), (void**)&rnd_state->dev_mults);
    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned long long), (void**)&rnd_state->dev_words);
    hipblasSetVector(NUM_RND_STREAMS, sizeof(unsigned int), host_mults, 1, rnd_state->dev_mults, 1);
    //hipMalloc((void **)&rnd_state->dev_mults, NUM_RND_STREAMS * sizeof(unsigned int));
    //hipMalloc((void **)&rnd_state->dev_words, NUM_RND_STREAMS * sizeof(unsigned long long));
    //hipMemcpy(rnd_state->dev_mults, host_mults, NUM_RND_STREAMS * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    kSeedRandom<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, seed);
 
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

/* ------------------------------ Utility routines ------------------------------ */

extern int get_leading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[1] : mat->size[0];
}

extern int get_nonleading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[0] : mat->size[1];
}

extern void set_transpose(cudamat* mat, int is_trans) {
    mat->is_trans = is_trans;
}

inline char get_transpose_char(cudamat* mat) {
    return mat->is_trans ? 't' : 'n';
}

extern void cuda_sync_threads() {
    hipDeviceSynchronize();
}

/* ------------------------------ Allocating/moving data ------------------------------ */

extern int allocate_device_memory(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    hipblasStatus_t stat;

    stat = cublasAlloc(len, sizeof(mat->data_device[0]), (void**)&mat->data_device);

    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}

extern int copy_to_host(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    if (mat->on_device) {
            hipblasGetVector(len, sizeof(mat->data_host[0]), mat->data_device, 1, mat->data_host, 1);

        if (check_cublas_error())
            return CUBLAS_ERROR;
    } else
       return ERROR_NOT_ON_DEVICE;
 
    return 0;
}

extern int copy_to_device(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(len, sizeof(mat->data_host[0]), mat->data_host, 1, mat->data_device, 1);
    
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

extern int copy_on_device(cudamat* mat1, cudamat* mat2) {
    int len = mat1->size[0]*mat1->size[1];

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasScopy(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

extern int get_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = source->size[0];
    int width = source->size[1];

    if ((end - start) != target->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    kGetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int set_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = target->size[0];
    int width = target->size[1];

    if ((end - start) != source->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    kSetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int copy_transpose(cudamat* source, cudamat* target) {
    unsigned int height = source->size[0];
    unsigned int width = source->size[1];

    if (source->size[0] != target->size[1] || source->size[1] != target->size[0])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    // setup execution parameters
    unsigned int grid_x = height / COPY_BLOCK_SIZE;
    if (height % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = width / COPY_BLOCK_SIZE;
    if (width % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, 1);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);

    kTranspose<<< grid, threads >>>(target->data_device, source->data_device, height, width);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int free_device_memory(cudamat* mat) {
    if (mat->owns_data && mat->on_device) {
        hipblasStatus_t stat;

        stat = cublasFree(mat->data_device);
        mat->on_device = 0;

        if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error())
            return CUBLAS_ERROR;
    }

    return 0;
}

extern int reshape(cudamat* mat, unsigned int m, unsigned int n) {
    if (mat->size[0] * mat->size[1] != m * n)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    mat->size[0] = m;
    mat->size[1] = n;

    return 0;
}

extern int get_slice(cudamat* source, cudamat* target, unsigned int first_col, unsigned int last_col) {
    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (last_col > source->size[1] || (first_col >= last_col))
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_col * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->size[0] = source->size[0];
    target->size[1] = last_col - first_col;
    target->is_trans = 0;
    target->owns_data = 0;

    return 0;
}

extern int get_vector_slice(cudamat* source, cudamat* target, unsigned int first_ind, unsigned int last_ind) {
    // source must be a vector
    if (source->size[0] > 1 && source->size[1] > 1)
        return ERROR_GENERIC;

    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (first_ind >= last_ind)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_ind * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->is_trans = 0;
    target->owns_data = 0;

    if (source->size[0] > 1) {
        if (last_ind > source->size[0])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = last_ind - first_ind;
        target->size[1] = 1;
    } else {
        if (last_ind > source->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = 1;
        target->size[1] = last_ind - first_ind;
    }

    return 0;
}

/* ------------------------------ Initialization routines ------------------------------ */

extern void init_from_array(cudamat* mat, float* data, int m, int n) {
    mat->data_host = data;
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 1;
    mat->is_trans = 0;
    mat->owns_data = 1;
}

extern int init_empty(cudamat* mat, int m, int n) {
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 0;
    mat->is_trans = 0;
    mat->owns_data = 1;

    return allocate_device_memory(mat);
}

/* ------------------------------ Random number generation ------------------------------ */
extern int fill_with_rand(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomUniform<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int fill_with_randn(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomGaussian<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
/* ------------------------------ Algebraic operations ------------------------------ */

extern int add_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError()) {
        return CUDA_ERROR;
    }

    return 0;
}

extern int add_col_mult(cudamat* mat, cudamat* vec, cudamat* target, float mult) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddColMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, mult, w, h);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int add_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultByColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultByRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int less_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int less_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int equals(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kEquals<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int equals_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kEqualsScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int max_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        kMaxColumnwise<<<w,32>>>(mat->data_device, target->data_device, w, h);

        if (SYNC_THREADS)
            hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int sign(cudamat* mat, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSign<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_sigmoid(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplySigmoid<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_tanh(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyTanh<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_abs(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyAbs<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_log_1_plus_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyLog1PlusExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_log(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLog<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_sqrt(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSqrt<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_pow(cudamat* mat, float pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kPow<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_pow_matrix(cudamat* mat, cudamat* pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kPowMatrix<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int reciprocal(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kReciprocal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int dot(cudamat* mat1, cudamat* mat2, cudamat* target, float beta, float alpha) {
    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (get_leading_dimension(mat1) != get_leading_dimension(target) ||
        get_nonleading_dimension(mat2) != get_nonleading_dimension(target) ||
        get_nonleading_dimension(mat1) != get_leading_dimension(mat2)) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    int m = get_leading_dimension(mat1),
        k = get_leading_dimension(mat2),
        n = get_nonleading_dimension(mat2);

    hipblasSgemm(get_transpose_char(mat1), get_transpose_char(mat2), 
                m, n, k,
                alpha, mat1->data_device, mat1->size[0],
                mat2->data_device, mat2->size[0],
                beta, target->data_device, target->size[0]);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    if (SYNC_THREADS) 
        hipDeviceSynchronize();

    return 0;
}

extern float vdot(cudamat* mat1, cudamat* mat2, int* err_code) {
    int len = mat1->size[0]*mat1->size[1];
    float res;

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans) {
        *err_code = ERROR_TRANSPOSEDNESS;
        return 0;
    }

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1]) { 
        *err_code = ERROR_INCOMPATIBLE_DIMENSIONS;
        return 0;
    }

    res = hipblasSdot(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}

/* Perform the operation mat1 = mat1 + alpha * mat2. mat1 and mat2 must
   have the same transposedness. */
extern int add_mult(cudamat* mat1, cudamat* mat2, float alpha) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasSaxpy(len, alpha, mat2->data_device, 1, mat1->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

extern int add_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat1 == target) {
        hipblasSaxpy(len, 1, mat2->data_device, 1, mat1->data_device, 1);
 
        if (check_cublas_error())
            return CUBLAS_ERROR;

    } else {
        kAdd<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);
 
        if (SYNC_THREADS)
            hipDeviceSynchronize();

        if (checkCUDAError())
            return CUDA_ERROR;
    }
 
     return 0;
}

extern int subtract_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSubtract<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int divide_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivide<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

/* Elementwise multiplication of 2 matrices */
extern int mult_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int assign_scalar(cudamat* mat, float alpha) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kAssignScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat == target) {
        hipblasSscal(len, alpha, mat->data_device, 1);
 
        if (check_cublas_error())
            return CUBLAS_ERROR;

    } else {
        kMultScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

        if (SYNC_THREADS) 
            hipDeviceSynchronize();

        if (checkCUDAError())
            return CUDA_ERROR;
    }
 
    return 0;
}

extern int divide_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivideScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int add_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern float euclid_norm(cudamat* mat, int* err_code) {
    int len = mat->size[0]*mat->size[1];

    float res =  hipblasSnrm2(len, mat->data_device, 1);

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}

extern int selectRows(cudamat* source, cudamat* target, cudamat* indices){
    const int nRetRows = indices->size[1];

    if (nRetRows==0) return 0;

    dim3 gridDim((nRetRows+31)/32);
    dim3 blockDim(32);

    kSelectRows<<<gridDim, blockDim>>>(source->data_device, target->data_device, indices->data_device, nRetRows, source->size[0], source->size[1]);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int setSelectedRows(cudamat* target, cudamat* source, cudamat* indices){
    const int nSetRows = indices->size[1];

    if (nSetRows==0)
        return 0;

    dim3 gridDim((nSetRows+31)/32);
    dim3 blockDim(32);

    kSetSelectedRows<<<gridDim, blockDim>>>(target->data_device, source->data_device, indices->data_device, nSetRows, target->size[0], target->size[1]);

    if (SYNC_THREADS)
        hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

}
